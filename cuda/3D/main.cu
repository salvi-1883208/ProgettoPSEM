// cuda implementation of the dla algorithm in 3 dimensions


#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>

// move the particle in the random direction
__device__ void move_particle(int* x, int* y, int* z, int m);

// atomi CAS for bool
static __inline__ __device__ bool atomicCAS(bool* address, bool compare, bool val);

// write the matrix to a file
int write_matrix_to_file(bool* matrix, int dim);

// kernel to set up the seed for each thread
__global__ void setup_kernel(hiprandState* state, int randomSeed) {
    // calculate thread id
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(randomSeed, id, 0, &state[id]);
}

// kernel to perform the dla algorithm
__global__ void dla_kernel(hipPitchedPtr d_grid, hiprandState* state, int gridSize, int maxIterations) {
    // calculate thread id
    // copy the random state to the local memory
    hiprandState localState = state[threadIdx.x + blockIdx.x * blockDim.x];

    // get the pointer to the grid
    bool* grid = (bool*)d_grid.ptr;
    // get the pitch of the grid
    size_t pitch = d_grid.pitch;
    // calculate the slice pitch (a 2D slice of the 3D grid)
    size_t slicePitch = d_grid.pitch * gridSize;

    // initialize the starting position of the particle
    int x;
    int y;
    int z;

    // initialize the counter for the number of iterations
    int g = 0;

    // if the particle has been generated on a stuck particle, generate a new position
    do {
        x = hiprand(&localState) % gridSize;
        y = hiprand(&localState) % gridSize;
        z = hiprand(&localState) % gridSize;
    } while (grid[x * slicePitch + y * pitch + z] && g++ < maxIterations);

    // iterate until the particle is attached to the grid or it did more than
    // maxIterations number of iterations
    while (g++ < maxIterations) {
        // if the particle is outside the grid, move it back inside
        x = min(max(x, 1), gridSize - 2);
        y = min(max(y, 1), gridSize - 2);
        z = min(max(z, 1), gridSize - 2);

        // if the particle is close to an already stuck particle
        for (int i = -1; i <= 1; i++)
            for (int j = -1; j <= 1; j++)
                for (int k = -1; k <= 1; k++)
                    if (grid[(x + k) * slicePitch + (y + j) * pitch + (z + i)]) {
                        // if the particle is close to an already stuck particle
                        // attach it to the grid
                        atomicCAS(&grid[x * slicePitch + y * pitch + z], 0, 1);
                        return;
                    }

        // calculate the random direction of the particle
        // and move the particle in the random direction
        move_particle(&x, &y, &z, hiprand(&localState) % 26);
    }

    // if the particle did more than MAX_ITER number of iterations, skip it

    // I have to do this because of warp divergence, if I remove this it won't work
    __syncwarp();

    return;
}

int main(int argc, char* argv[]) {
    // command line input: grid size, number of particles, number of steps, seed
    // coordinates, block size, random seed
    if ((argc) < 7) {
        printf(
            "Arguments are: square grid size, number of particles times block "
            "size, number of maximum steps, seed coordinates, the number of "
            "threads per block, seed for the curand() function.\n");
        return -1;
    }

    // get grid size from args
    int gridSize = atoi(argv[1]) | 1;

    // get number of particles sent from args
    int numParticles = atoi(argv[2]);

    // get number of iterations for each particle from args
    int maxIterations = atoi(argv[3]);

    // get seed coordinates from args
    int si = atoi(argv[4]) - 1;
    int sj = atoi(argv[5]) - 1;
    int sk = atoi(argv[6]) - 1;

    // if given out image coordinates place seed in the middle
    if (si < 0 || sj < 0 || sk < 0 || si > gridSize || sj > gridSize || sk > gridSize) {
        printf("Given outside of image seed coordinates.\n");
        printf("Setting seed coordinates to %d, %d, %d.\n", gridSize / 2, gridSize / 2, gridSize / 2);
        si = (gridSize - 1) / 2;
        sj = (gridSize - 1) / 2;
        sk = (gridSize - 1) / 2;
    }

    // get number of threads per block from args
    int blockSize;
    if (argc >= 8)
        blockSize = atoi(argv[7]);
    else
        blockSize = 1024;  // I am using a 1080, so I can use a maximum of 1024
                           // threads per block

    // calculate the number of particles based on the number of threads per block
    numParticles *= blockSize;

    // if the random seed is given from the command line arguments
    int randomSeed;
    if (argc == 9)
        // get seed for the rand() function from args
        randomSeed = atoi(argv[8]);
    else
        // if the random seed is not given from the command line arguments, use a
        // default value
        randomSeed = 3521;

    // calculate the number of blocks
    int blocks = (numParticles + blockSize - 1) / blockSize;

    // allocate the grid on the host
    bool* h_grid = (bool*)malloc(gridSize * gridSize * gridSize * sizeof(bool));

    // initialize the grid
    for (int i = 0; i < gridSize; i++)
        for (int j = 0; j < gridSize; j++)
            for (int k = 0; k < gridSize; k++)
                h_grid[i * gridSize * gridSize + j * gridSize + k] = 0;

    // place the seed in si, sj
    h_grid[si * gridSize * gridSize + sj * gridSize + sk] = 1;

    // allocate the grid in the device memory (aligned)
    hipExtent extent = make_hipExtent(gridSize, gridSize, gridSize);
    hipPitchedPtr d_grid;
    hipMalloc3D(&d_grid, extent);

    // copy the grid from the host to the device
    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void*)h_grid, gridSize * sizeof(bool), gridSize, gridSize);
    copyParams.dstPtr = d_grid;
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    // allocate the array of the random states in the device memory
    hiprandState* d_state;
    hipMalloc((void**)&d_state, blocks * blockSize * sizeof(hiprandState));

    printf("\nSimulating growth...\n");

    // time execution start
    float time;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // launch the kernel to set up the seed for each thread
    setup_kernel<<<blocks, blockSize>>>(d_state, randomSeed);

    // wait for the kernel to finish
    hipDeviceSynchronize();

    // launch the kernel to perform the dla algorithm
    dla_kernel<<<blocks, blockSize>>>(d_grid, d_state, gridSize, maxIterations);

    // wait for the kernel to finish
    hipDeviceSynchronize();

    // stop timer for execution time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Simulation finished.\n\n");

    // copy the grid from the device to the host
    hipMemcpy3DParms copyParams2 = {0};
    copyParams2.srcPtr = d_grid;
    copyParams2.dstPtr = make_hipPitchedPtr((void*)h_grid, gridSize * sizeof(bool), gridSize, gridSize);
    copyParams2.extent = extent;
    copyParams2.kind = hipMemcpyDeviceToHost;
    hipMemcpy3D(&copyParams2);

    // save the grid as a txt file and get the number of stuck particles
    int stuck = write_matrix_to_file(h_grid, gridSize);

    // print the number of skipped particles
    printf("Of %d particles:\n - drawn %d,\n - skipped %d.\n\n", numParticles, stuck, numParticles - stuck);

    // print the time to simulate in seconds
    printf("Execution time in seconds: %f\n", time / 1000);

    // free the memory
    hipFree(h_grid);
    hipFree(d_state);

    return 0;
}

// save the grid to a file
int write_matrix_to_file(bool* matrix, int dim) {
    int count = 0;
    FILE* fp = fopen("matrix.txt", "w");
    if (fp == NULL) {
        printf("Error opening file %s\n", "matrix.txt");
        exit(1);
    }
    fprintf(fp, "%d\n", dim);

    for (int i = 0; i < dim; i++)
        for (int j = 0; j < dim; j++)
            for (int k = 0; k < dim; k++)
                if (matrix[(i * dim * dim) + (j * dim) + k]) {
                    fprintf(fp, "%d %d %d\n", i, j, k);
                    count++;
                }

    fclose(fp);

    return count;
}

// define the offsets for the 26 directions
__constant__ int dx[26] = {-1, -1, -1, -1, -1, -1, -1, -1, -1, 0, 0, 0, 0,
                           0, 1, 1, 1, 1, 1, 1, 1, 1, 1, 0, 0, 0};
__constant__ int dy[26] = {-1, -1, -1, 0, 0, 0, 1, 1, 1, -1, -1, -1, 0,
                           0, -1, -1, -1, 0, 0, 0, 1, 1, 1, 1, 1, 1};
__constant__ int dz[26] = {-1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1,
                           1, -1, 0, 1, -1, 0, 1, -1, 0, 1, -1, 0, 1};

// move the particle in the random direction (out of 26)
__device__ void move_particle(int* x, int* y, int* z, int m) {
    (*x) += dx[m];
    (*y) += dy[m];
    (*z) += dz[m];
}

// copied from stackoverflow
static __inline__ __device__ bool atomicCAS(bool* address, bool compare, bool val) {
    unsigned long long addr = (unsigned long long)address;
    unsigned pos = addr & 3;             // byte position within the int
    int* int_addr = (int*)(addr - pos);  // int-aligned address
    int old = *int_addr, assumed, ival;

    bool current_value;

    do {
        current_value = (bool)(old & ((0xFFU) << (8 * pos)));

        if (current_value != compare)  // If we expected that bool to be different, then
            break;                     // stop trying to update it and just return it's current value

        assumed = old;
        if (val)
            ival = old | (1 << (8 * pos));
        else
            ival = old & (~((0xFFU) << (8 * pos)));
        old = atomicCAS(int_addr, assumed, ival);
    } while (assumed != old);

    return current_value;
}